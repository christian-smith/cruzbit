#include "hip/hip_runtime.h"
// mine.cu
// 19-Jul-19 Provides cgo hooks to manage mining on Nvidia devices -asdvxgxasjab

#include "sha3.h"
#include "sha3_cu.h"
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>
#include <string.h>

static const char *_cudaErrorToString(hipError_t error) {
  switch (error) {
  case hipSuccess:
    return "hipSuccess";

  case hipErrorMissingConfiguration:
    return "hipErrorMissingConfiguration";

  case hipErrorOutOfMemory:
    return "hipErrorOutOfMemory";

  case hipErrorNotInitialized:
    return "hipErrorNotInitialized";

  case hipErrorLaunchFailure:
    return "hipErrorLaunchFailure";

  case hipErrorPriorLaunchFailure:
    return "hipErrorPriorLaunchFailure";

  case hipErrorLaunchTimeOut:
    return "hipErrorLaunchTimeOut";

  case hipErrorLaunchOutOfResources:
    return "hipErrorLaunchOutOfResources";

  case hipErrorInvalidDeviceFunction:
    return "hipErrorInvalidDeviceFunction";

  case hipErrorInvalidConfiguration:
    return "hipErrorInvalidConfiguration";

  case hipErrorInvalidDevice:
    return "hipErrorInvalidDevice";

  case hipErrorInvalidValue:
    return "hipErrorInvalidValue";

  case hipErrorInvalidPitchValue:
    return "hipErrorInvalidPitchValue";

  case hipErrorInvalidSymbol:
    return "hipErrorInvalidSymbol";

  case hipErrorMapFailed:
    return "hipErrorMapFailed";

  case hipErrorUnmapFailed:
    return "hipErrorUnmapFailed";

  case cudaErrorInvalidHostPointer:
    return "cudaErrorInvalidHostPointer";

  case hipErrorInvalidDevicePointer:
    return "hipErrorInvalidDevicePointer";

  case hipErrorInvalidTexture:
    return "hipErrorInvalidTexture";

  case cudaErrorInvalidTextureBinding:
    return "cudaErrorInvalidTextureBinding";

  case hipErrorInvalidChannelDescriptor:
    return "hipErrorInvalidChannelDescriptor";

  case hipErrorInvalidMemcpyDirection:
    return "hipErrorInvalidMemcpyDirection";

  case cudaErrorAddressOfConstant:
    return "cudaErrorAddressOfConstant";

  case cudaErrorTextureFetchFailed:
    return "cudaErrorTextureFetchFailed";

  case cudaErrorTextureNotBound:
    return "cudaErrorTextureNotBound";

  case cudaErrorSynchronizationError:
    return "cudaErrorSynchronizationError";

  case cudaErrorInvalidFilterSetting:
    return "cudaErrorInvalidFilterSetting";

  case cudaErrorInvalidNormSetting:
    return "cudaErrorInvalidNormSetting";

  case cudaErrorMixedDeviceExecution:
    return "cudaErrorMixedDeviceExecution";

  case hipErrorDeinitialized:
    return "hipErrorDeinitialized";

  case hipErrorUnknown:
    return "hipErrorUnknown";

  case cudaErrorNotYetImplemented:
    return "cudaErrorNotYetImplemented";

  case cudaErrorMemoryValueTooLarge:
    return "cudaErrorMemoryValueTooLarge";

  case hipErrorInvalidHandle:
    return "hipErrorInvalidHandle";

  case hipErrorNotReady:
    return "hipErrorNotReady";

  case hipErrorInsufficientDriver:
    return "hipErrorInsufficientDriver";

  case hipErrorSetOnActiveProcess:
    return "hipErrorSetOnActiveProcess";

  case cudaErrorInvalidSurface:
    return "cudaErrorInvalidSurface";

  case hipErrorNoDevice:
    return "hipErrorNoDevice";

  case hipErrorECCNotCorrectable:
    return "hipErrorECCNotCorrectable";

  case hipErrorSharedObjectSymbolNotFound:
    return "hipErrorSharedObjectSymbolNotFound";

  case hipErrorSharedObjectInitFailed:
    return "hipErrorSharedObjectInitFailed";

  case hipErrorUnsupportedLimit:
    return "hipErrorUnsupportedLimit";

  case cudaErrorDuplicateVariableName:
    return "cudaErrorDuplicateVariableName";

  case cudaErrorDuplicateTextureName:
    return "cudaErrorDuplicateTextureName";

  case cudaErrorDuplicateSurfaceName:
    return "cudaErrorDuplicateSurfaceName";

  case cudaErrorDevicesUnavailable:
    return "cudaErrorDevicesUnavailable";

  case hipErrorInvalidImage:
    return "hipErrorInvalidImage";

  case hipErrorNoBinaryForGpu:
    return "hipErrorNoBinaryForGpu";

  case cudaErrorIncompatibleDriverContext:
    return "cudaErrorIncompatibleDriverContext";

  case hipErrorPeerAccessAlreadyEnabled:
    return "hipErrorPeerAccessAlreadyEnabled";

  case hipErrorPeerAccessNotEnabled:
    return "hipErrorPeerAccessNotEnabled";

  case hipErrorContextAlreadyInUse:
    return "hipErrorContextAlreadyInUse";

  case hipErrorProfilerDisabled:
    return "hipErrorProfilerDisabled";

  case hipErrorProfilerNotInitialized:
    return "hipErrorProfilerNotInitialized";

  case hipErrorProfilerAlreadyStarted:
    return "hipErrorProfilerAlreadyStarted";

  case hipErrorProfilerAlreadyStopped:
    return "hipErrorProfilerAlreadyStopped";

  case hipErrorAssert:
    return "hipErrorAssert";

  case cudaErrorTooManyPeers:
    return "cudaErrorTooManyPeers";

  case hipErrorHostMemoryAlreadyRegistered:
    return "hipErrorHostMemoryAlreadyRegistered";

  case hipErrorHostMemoryNotRegistered:
    return "hipErrorHostMemoryNotRegistered";

  case hipErrorOperatingSystem:
    return "hipErrorOperatingSystem";

  case hipErrorPeerAccessUnsupported:
    return "hipErrorPeerAccessUnsupported";

  case cudaErrorLaunchMaxDepthExceeded:
    return "cudaErrorLaunchMaxDepthExceeded";

  case cudaErrorLaunchFileScopedTex:
    return "cudaErrorLaunchFileScopedTex";

  case cudaErrorLaunchFileScopedSurf:
    return "cudaErrorLaunchFileScopedSurf";

  case cudaErrorSyncDepthExceeded:
    return "cudaErrorSyncDepthExceeded";

  case cudaErrorLaunchPendingCountExceeded:
    return "cudaErrorLaunchPendingCountExceeded";

  case cudaErrorNotPermitted:
    return "cudaErrorNotPermitted";

  case hipErrorNotSupported:
    return "hipErrorNotSupported";

  case cudaErrorHardwareStackError:
    return "cudaErrorHardwareStackError";

  case cudaErrorIllegalInstruction:
    return "cudaErrorIllegalInstruction";

  case cudaErrorMisalignedAddress:
    return "cudaErrorMisalignedAddress";

  case cudaErrorInvalidAddressSpace:
    return "cudaErrorInvalidAddressSpace";

  case cudaErrorInvalidPc:
    return "cudaErrorInvalidPc";

  case hipErrorIllegalAddress:
    return "hipErrorIllegalAddress";

  case hipErrorInvalidKernelFile:
    return "hipErrorInvalidKernelFile";

  case hipErrorInvalidGraphicsContext:
    return "hipErrorInvalidGraphicsContext";

  case cudaErrorStartupFailure:
    return "cudaErrorStartupFailure";

  case cudaErrorApiFailureBase:
    return "cudaErrorApiFailureBase";

  case cudaErrorNvlinkUncorrectable:
    return "cudaErrorNvlinkUncorrectable";

  case cudaErrorJitCompilerNotFound:
    return "cudaErrorJitCompilerNotFound";

  case hipErrorCooperativeLaunchTooLarge:
    return "hipErrorCooperativeLaunchTooLarge";
  }

  return "<unknown>";
}

__device__ int memcmp_cu(const void *p1, const void *p2, size_t len) {
  for (size_t i = 0; i < len; i++) {
    uint8_t b1 = ((uint8_t *)p1)[i];
    uint8_t b2 = ((uint8_t *)p2)[i];
    if (b1 < b2) {
      return -1;
    }
    if (b1 > b2) {
      return 1;
    }
  }
  return 0;
}

__device__ int strlen_cu(char *s) {
  int i;
  for (i = 0; s[i] != '\0';) {
    i++;
  }
  return i;
}

__device__ char *reverse(char *str) {
  char tmp, *src, *dst;
  size_t len;
  if (str != NULL) {
    len = strlen_cu(str);
    if (len > 1) {
      src = str;
      dst = src + len - 1;
      while (src < dst) {
        tmp = *src;
        *src++ = *dst;
        *dst-- = tmp;
      }
    }
  }
  return str;
}

__device__ int itoa(int64_t n, char s[]) {
  int i;
  int64_t sign;

  if ((sign = n) < 0) /* record sign */
    n = -n;           /* make n positive */
  i = 0;

  do {                     /* generate digits in reverse order */
    s[i++] = n % 10 + '0'; /* get next digit */
  } while ((n /= 10) > 0); /* delete it */

  if (sign < 0)
    s[i++] = '-';

  s[i] = '\0';
  reverse(s);
  return i;
}

__device__ void debug_print_buf(const void *buf, size_t len) {
  for (int i = 0; i < len; i++) {
    printf("%c", ((char *)buf)[i]);
  }
  printf("\n");
}

__device__ void debug_print_hash(const void *hash) {
  for (int i = 0; i < 32; i++) {
    printf("%02x", ((char *)hash)[i] & 0xFF);
  }
  printf("\n");
}

// called by each device thread
__global__ void try_solve(int64_t start_nonce, const sha3_ctx_t *prev_sha3,
                          const void *last, size_t last_len, const void *target,
                          int64_t *good_nonce) {
  uint8_t hash[32];
  uint8_t nonce_s[20];

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int64_t nonce = start_nonce + (int64_t)index;
  size_t n = (size_t)itoa(nonce, (char *)nonce_s);

  sha3_ctx_t sha3;
  memcpy(&sha3, prev_sha3, sizeof(sha3_ctx_t));
  sha3_update_cu(&sha3, nonce_s, n);
  sha3_update_cu(&sha3, last, last_len);
  sha3_final_cu(hash, &sha3);

  if (memcmp_cu(hash, target, 32) <= 0) {
    // found a solution. not thread-safe but a race is very unlikely
    *good_nonce = nonce;
  }
}

// device-local state
struct miner_state {
  int num_blocks, block_size, max_threads;
  sha3_ctx_t *prev_sha3_cu;
  void *last_cu, *target_cu;
  size_t last_len;
  int64_t *nonce_cu;
};

static struct miner_state *states = 0;

extern "C" {

// called on startup
int cuda_init() {
  int device_count = -1;
  hipError_t error = hipGetDeviceCount(&device_count);
  if (error != hipSuccess) {
    printf("hipGetDeviceCount: %s\n", _cudaErrorToString(error));
    return -1;
  }
  if (device_count <= 0) {
    return -1;
  }

  states = new struct miner_state[device_count];

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t props;
    error = hipGetDeviceProperties(&props, i);
    if (error != hipSuccess) {
      printf("hipGetDeviceProperties: %s\n", _cudaErrorToString(error));
      return -1;
    }

    states[i].max_threads =
        props.maxThreadsPerMultiProcessor * props.multiProcessorCount;
    states[i].block_size = props.warpSize;
    states[i].num_blocks = states[i].max_threads / states[i].block_size;

    error = hipSetDevice(i);
    if (error != hipSuccess) {
      printf("hipSetDevice: %s\n", _cudaErrorToString(error));
      return -1;
    }

    error = hipDeviceReset();
    if (error != hipSuccess) {
      printf("hipDeviceReset: %s\n", _cudaErrorToString(error));
      return -1;
    }

#if 0
    // I tried this but it noticeably impacted performance
    error = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (error != hipSuccess) {
      printf("hipSetDeviceFlags: %s\n", _cudaErrorToString(error));
      return -1;
    }
#endif

    // allocate memory used on device written to by the host
    hipMalloc(&states[i].prev_sha3_cu, sizeof(sha3_ctx_t));
    hipMalloc(&states[i].last_cu, 512);
    hipMalloc(&states[i].target_cu, 32);
    hipMalloc(&states[i].nonce_cu, sizeof(int64_t));
  }

  return device_count;
}

// called after updating the block header
int miner_update(int miner_num, const void *first, size_t first_len,
                 const void *last, size_t last_len, const void *target) {
  hipSetDevice(miner_num);

  // hash the first (largest) part of the header once and copy the state
  sha3_ctx_t sha3;
  sha3_init(&sha3, 32);
  sha3_update(&sha3, first, first_len);
  hipMemcpy(states[miner_num].prev_sha3_cu, &sha3, sizeof(sha3_ctx_t),
             hipMemcpyHostToDevice);

  // copy the end part of the header
  states[miner_num].last_len = last_len;
  hipMemcpy(states[miner_num].last_cu, last, last_len, hipMemcpyHostToDevice);

  // copy the target
  hipMemcpy(states[miner_num].target_cu, target, 32, hipMemcpyHostToDevice);

  // set the nonce to "not found"
  hipMemset(states[miner_num].nonce_cu, 0x7F, sizeof(int64_t));
  hipMemset(states[miner_num].nonce_cu, 0xFF, sizeof(int64_t) - 1);

  return states[miner_num].num_blocks * states[miner_num].block_size;
}

// called in a loop until solved
// returns a solving nonce if found; otherwise 0x7FFFFFFFFFFFFFFF
int64_t miner_mine(int miner_num, int64_t start_nonce) {
  hipSetDevice(miner_num);
  int64_t nonce;
  int num_blocks = states[miner_num].num_blocks;
  int block_size = states[miner_num].block_size;
  try_solve<<<num_blocks, block_size>>>(
      start_nonce, states[miner_num].prev_sha3_cu, states[miner_num].last_cu,
      states[miner_num].last_len, states[miner_num].target_cu,
      states[miner_num].nonce_cu);
  hipDeviceSynchronize();
  hipMemcpy(&nonce, states[miner_num].nonce_cu, sizeof(int64_t),
             hipMemcpyDeviceToHost);
  return nonce;
}
}
